#include "hip/hip_runtime.h"
#pragma once
#include ""

#include "../Header files/metric.cuh"
#include "../Header files/vector_operations.cuh"
#include "../Header files/Constants.cuh"

#include "../../C++/Header files/Code.h"
#include "../../C++/Header files/IntegrationDefines.h"



namespace metric {
#ifndef __CUDA_ARCH__
	const double BUTCHER_TABLEAU;
	const double BUTCHER_ERROR;
#else
	__device__ __constant__ const double BUTCHER_TABLEAU;
	__device__ __constant__ const double BUTCHER_ERROR;
#endif // !__CUDA_ARCH__

	/// <summary>
	/// Sets parameters of the metric required for integration
	/// </summary>
	/// <param name="afactor"></param> The factor of angular momentum over mass
	/// <param name="accretionRadius"></param> The maximum radius of the accretion disk 
	/// <returns></returns>
	template <class T> __host__ void setMetricParameters(T afactor, T accretionRadius, bool useDisk) {
		metric::a<T> = afactor;
		metric::asq<T> = afactor * afactor;
		metric::accretionDiskRadius<T> = accretionRadius;
		metric::useAccretionDisk = useDisk;

		hipMemcpyToSymbol(HIP_SYMBOL(metric::a_dev<T>), &a<T>, sizeof(T));
		hipMemcpyToSymbol(HIP_SYMBOL(metric::asq_dev<T>), &asq<T>, sizeof(T));
		hipMemcpyToSymbol(HIP_SYMBOL(metric::accretionDiskRadius_dev<T>), &accretionDiskRadius<T>, sizeof(T));
		hipMemcpyToSymbol(HIP_SYMBOL(metric::useAccretionDisk_dev), &useAccretionDisk, sizeof(bool));
	}

	template <class T>
	__device__ __host__ __forceinline__ T sq(T x) {
		return x * x;
	}

	template <class T>
	__device__ __host__ __forceinline__ T sq3(T x) {
		return x * x * x;
	}

	template <class T> __device__ __host__ __forceinline__ T _Delta(T r, T rsq) {
		return rsq - 2. * r + BH_ASQ;
	};

	template <class T> __device__ __host__ __forceinline__ T _Sigma(T r, T theta, T rsq, T sinsq) {
		return sqrt(sq(rsq + BH_ASQ) - BH_ASQ * _Delta(r, rsq) * sinsq);
	};

	template <class T> __device__ __host__ __forceinline__ T _ro(T r, T theta, T rsq, T cossq) {
		return sqrt(rsq + BH_ASQ * cossq);
	};

	template <class T> __device__ __host__ __forceinline__ T _rosq(T r, T theta, T rsq, T cossq) {
		return rsq + BH_ASQ * cossq;
	};

	template <class T> __device__ __host__ __forceinline__ T _w(T r, T theta, T rsq, T sinsq) {
		return 2. * BH_A * r / sq(_Sigma(r, theta, rsq, sinsq));
	};

	template <class T> __device__ __host__  __forceinline__ T _wbar(T r, T theta, T rsq, T sinsq, T cossq) {
		return _Sigma(r, theta, rsq, sinsq) * sin(theta) / _ro(r, theta, rsq, cossq);
	};

	template <class T> __device__ __host__ __forceinline__  T _alpha(T r, T theta, T rsq, T sinsq, T cossq) {
		return _ro(r, theta, rsq, cossq) * sqrt(_Delta(r, rsq)) / _Sigma(r, theta, rsq, sinsq);
	};

	template <class T> __device__ __host__ __forceinline__ T _P(T r, T b, T rsq) {
		return rsq + BH_ASQ - BH_A * b;
	}

	template <class T> __device__ __host__ __forceinline__ T _R(T r, T theta, T b, T q, T rsq) {
		return sq(_P(r, b, rsq)) - _Delta(r, rsq) * (sq((b - BH_A)) + q);
	};

	template <class T> __device__ __host__ __forceinline__ T _BigTheta(T r, T theta, T b, T q, T sinsq, T cossq, T bsq) {
		return q - cossq * (bsq / sinsq - BH_ASQ);
	};

	template <class T> __device__ __host__ __forceinline__ T _gtt_theta_half_pi(T r) {
		return -(1-(2/r));
	};

	template <class T> __device__ __host__ __forceinline__ T _gphiphi_theta_half_pi(T r, T rsq) {
		return rsq+BH_ASQ + ((2*BH_ASQ)/r);
	};

	template <class T> __device__ __host__ __forceinline__ T _gtphi_theta_half_pi(T r) {
		return  -2 * BH_A / r;
	};

	template <class T> __device__ __host__ __forceinline__ T _Omega(T r) {
		return 1. / (BH_A + pow(r, 1.5));
	};


	/// <summary>
	/// Calculates the gravitational redshift of a photon emited by a particle in a  orbit around the black hole at theta = 0.5pi as seen by a distant observer
	/// According to "Gravitational redshift in Kerr field" by Anuj Kumar Dubey and Asoke Kumar Sen
	/// </summary>
	/// <typeparam name="T"></typeparam>
	/// <param name="r"></param>
	/// <param name="theta"></param>
	/// <param name="rsq"></param>
	/// <param name="cossq"></param>
	/// <param name="sinsq"></param>
	/// <returns></returns>
	template <class T> __device__ __host__ __forceinline__ T calculate_gravitational_redshift(T r, T rsq) {
		return sqrt(- _gtt_theta_half_pi(r) - 2 * _gtphi_theta_half_pi(r) * _Omega(r) - _gphiphi_theta_half_pi(r, rsq) * sq(_Omega(r)));
	};


	template <class T> __device__ __host__ __forceinline__ T calcSpeed(T r, T theta) {
		T rsq = sq(r);
		T sp = _wbar(r, theta, rsq, sq(sin(theta)), sq(cos(theta))) / _alpha(r, theta, rsq, sq(sin(theta)), sq(cos(theta))) * (_Omega(r) - _w(r, theta, rsq, sq(sin(theta))));
		return sp;
	}

	template <class T> __host__ T findMinGoldSec(T theta, T bval, T qval, T ax, T b, T tol) {
		T gr = (sqrt(5.0) + 1.) / 2.;
		T c = b - (b - ax) / gr;
		T d = ax + (b - ax) / gr;
		while (fabs(c - d) > tol) {
			if (_R(c, theta, bval, qval, sq(c)) < _R(d, theta, bval, qval, sq(d))) {
				b = d;
			}
			else {
				ax = c;
			}
			c = b - (b - ax) / gr;
			d = ax + (b - ax) / gr;
		}
		return (ax + b) / 2;
	}


	template <class T> __host__ T checkRup(T rV, T thetaV, T bV, T qV) {
		if (BH_A == 0) return false;
		T min = findMinGoldSec(thetaV, bV, qV, rV, 4 * rV, 0.00001);
		return (_R(min, thetaV, bV, qV, sq(min)) >= 0);
	}

	template <class T> __host__ T _b0(T r0) {
		return -(sq3(r0) - 3. * sq(r0) + BH_ASQ * r0 + BH_ASQ) / (BH_A * (r0 - 1.));
	};

	template <class T> __host__ T _b0diff(T r0) {
		return (BH_ASQ + BH_A - 2. * r0 * (sq(r0) - 3. * r0 + 3.)) / (BH_A * sq(r0 - 1.));
	};

	template <class T> __host__ T _q0(T r0) {
		return -sq3(r0) * (sq3(r0) - 6. * sq(r0) + 9. * r0 - 4. * BH_ASQ) / (BH_ASQ * sq(r0 - 1.));
	};



	template <class T> __host__ T checkB_Q(T bV, T qV) {
		T _r1 = 2. * (1. + cos(2. * acos(-BH_A) / 3.));
		T _r2 = 2. * (1. + cos(2. * acos(BH_A) / 3.));
		T error = 0.0000001;
		T r0V = 2.0;
		T bcheck = 100;

		while (fabs(bV - bcheck) > error) {
			bcheck = _b0(r0V);
			T bdiffcheck = _b0diff(r0V);
			T rnew = r0V - (bcheck - bV) / bdiffcheck;
			if (rnew < 1) {
				r0V = 1.0001;
			}
			else {
				r0V = rnew;
			}
		}
		T qb = _q0(r0V);
		T _b1 = _b0(_r2);
		T _b2 = _b0(_r1);
		return ((_b1 >= bV) || (_b2 <= bV) || (qV >= qb));
	}

	template <class T> __host__ bool checkCelest(T pRV, T rV, T thetaV, T bV, T qV) {
		bool check1 = checkB_Q(bV, qV);
		bool check2 = !check1 && (pRV < 0);
		bool check3 = check1 && checkRup(rV, thetaV, bV, qV);
		return check2 || check3;
	}


	//Calculates derivatives of geodesics with the parameters given in var
	template <class T> __device__ __host__ void derivs(volatile T* var, volatile T* varOut, T b, T q) {
		T cosv = cos(thetaVar);
		T sinv = sin(thetaVar);
		T cossq = metric::sq(cosv);
		T sinsq = metric::sq(sinv);
		T bsq = metric::sq(b);
		T rsq = metric::sq(rVar);
		T delta = metric::_Delta(rVar, rsq);
		T rosq = metric::_rosq(rVar, thetaVar, rsq, cossq);
		T P = metric::_P(rVar, b, rsq);
		T prsq = metric::sq(pRVar);
		T pthetasq = metric::sq(pThetaVar);
		T R = metric::_R(rVar, thetaVar, b, q, rsq);
		T partR = (q + metric::sq(BH_A - b));
		T btheta = metric::_BigTheta(rVar, thetaVar, b, q, sinsq, cossq, bsq);
		T rosqsq = metric::sq(2 * rosq);
		T sqrosqdel = (metric::sq(rosq) * delta);
		T asqcossin = BH_ASQ * cosv * sinv;
		T rtwo = 2 * rVar - 2;

		drdz = delta / rosq * pRVar;
		dtdz = 1. / rosq * pThetaVar;
		dpdz = (2 * BH_A * P - (2 * BH_A - 2 * b) * delta + (2 * b * cossq * delta) / sinsq) / (rosq * 2 * delta);
		dprdz = (rtwo * btheta - rtwo * partR + 4 * rVar * P) / (rosq * (2 * delta)) - (prsq * rtwo) / (2 * rosq)
			+ (4 * pthetasq * rVar) / rosqsq - ((4 * rVar - 4) * (btheta * (delta)+R)) / (rosq * metric::sq(2 * delta))
			+ (4 * prsq * rVar * (delta)) / rosqsq - (rVar * (btheta * delta + R)) / sqrosqdel;
		dptdz = ((2 * cosv * sinv * (bsq / sinsq - BH_ASQ) + (2 * bsq * metric::sq3(cosv)) / metric::sq3(sinv)) * delta) /
			(rosq * 2 * delta) - (4 * asqcossin * pthetasq) / rosqsq - (4 * asqcossin * prsq * delta) /
			rosqsq + (asqcossin * (btheta * delta + R)) / sqrosqdel;
	}

	//Wraps the theta and phi coordinates back to their respective domains [0,PI] and [0,2PI) respectively returns wheter phi has been reduced back from larger than Pi.
	template <class T>  __device__ __host__ bool wrapToPi(T& thetaW, T& phiW) {
		bool ret = false;

		thetaW = fmodf(thetaW, PI2);
		if (thetaW < 0) {
			thetaW += PI2;
		}

		if (thetaW > PI) {
			thetaW -= 2 * (thetaW - PI);
			phiW += PI;
			ret = true;
		}
		phiW = fmodf(phiW, PI2);
		if (phiW < 0) {
			phiW += PI2;
		}

		return ret;
	}

	//Wraps the theta and phi coordinates back to their respective domains [0,PI] and [0,2PI) respectively returns wheter phi has been reduced back from larger than Pi.
	template <class T>  __device__ __host__ bool wrapPhiToPi(T & phiW) {
		bool ret = false;
		phiW += PI2;
		phiW = fmodf(phiW, PI2);
		return ret;
	}

	/// <summary>
	/// Calculates next coordinates for a given stepsize according to butcher tableau
	/// </summary>
	/// <param name="var">Position to step</param>
	/// <param name="dvdz">Derivatives at current position</param>
	/// <param name="h">Stepsize</param>
	/// <param name="varOut">output of next position</param>
	/// <param name="varErr">the estimated error of this step</param>
	/// <param name="b"></param>
	/// <param name="q"></param>
	/// <param name="aks"></param>
	/// <param name="varTmpInt"></param>
	/// <returns></returns>
	template <class T> __device__ __host__ static void rkck(volatile T* var, volatile T* dvdz, const T h,
		volatile T* varOut, volatile T* varErr, const T b, const T q, volatile T* aks,
		volatile T* varTmpInt) {
		int i;
		for (i = 0; i < NUMBER_OF_EQUATIONS; i++)
			varTmpInt[i] = var[i] + b21 * h * dvdz[i];
		metric::derivs(varTmpInt, aks, b, q);
		for (i = 0; i < NUMBER_OF_EQUATIONS; i++)
			varTmpInt[i] = var[i] + h * (b31 * dvdz[i] + b32 * aks[i]);
		metric::derivs(varTmpInt, (aks + 5), b, q);
		for (i = 0; i < NUMBER_OF_EQUATIONS; i++)
			varTmpInt[i] = var[i] + h * (b41 * dvdz[i] + b42 * aks[i] + b43 * aks[i + 5]);
		metric::derivs(varTmpInt, (aks + 10), b, q);
		for (i = 0; i < NUMBER_OF_EQUATIONS; i++)
			varTmpInt[i] = var[i] + h * (b51 * dvdz[i] + b52 * aks[i] + b53 * aks[i + 5] + b54 * aks[i + 10]);
		metric::derivs(varTmpInt, aks + 15, b, q);
		for (i = 0; i < NUMBER_OF_EQUATIONS; i++)
			varTmpInt[i] = var[i] + h * (b61 * dvdz[i] + b62 * aks[i] + b63 * aks[i + 5] + b64 * aks[i + 10] + b65 * aks[i + 15]);
		metric::derivs(varTmpInt, aks + 20, b, q);
		for (i = 0; i < NUMBER_OF_EQUATIONS; i++)
			varOut[i] = var[i] + h * (c1 * dvdz[i] + c3 * aks[i + 5] + c4 * aks[i + 10] + c6 * aks[i + 20]);
		for (i = 0; i < NUMBER_OF_EQUATIONS; i++)
			varErr[i] = h * (dc1 * dvdz[i] + dc3 * aks[i + 5] + dc4 * aks[i + 10] + dc5 * aks[i + 15] + dc6 * aks[i + 20]);
	}

	/// <summary>
	/// Steps the RK integrator with stepsize h and updates the step size
	/// </summary>
	/// <param name="var">Array with current variables</param>
	/// <param name="dvdz">Array with derivatives at current position</param>
	/// <param name="z">Sum of all steps taken</param>
	/// <param name="h">Current stepsize</param>
	/// <param name="varScal"></param>
	/// <param name="b"></param>
	/// <param name="q"></param>
	/// <param name="varErr"></param>
	/// <param name="varTemp">Array to use for temporary variable storage</param>
	/// <param name="aks"></param>
	/// <param name="varTmpInt"></param>
	/// <returns></returns>
	template <class T> __device__ __host__ static void rkqs(volatile T* var, volatile  T* dvdz, T& z, T& h,
		volatile T* varScal, const T b, const T q,
		volatile T* varErr, volatile T* varTemp, volatile T* aks, volatile T* varTmpInt) {

		rkck(var, dvdz, h, varTemp, varErr, b, q, aks, varTmpInt);
		T errmax = 0.0;
		for (int i = 0; i < NUMBER_OF_EQUATIONS; i++) errmax = fmax(errmax, fabs(varErr[i] / varScal[i]));
		errmax /= MIN_ACCURACY;
		if (errmax <= 1.0) {
			z += h;
			for (int i = 0; i < NUMBER_OF_EQUATIONS; i++) var[i] = varTemp[i];
			if (errmax > ERRCON) h = SAFETY * h * pow(errmax, PGROW);
			else h = ADAPTIVE * h;
		}
		else {
			h = fmin(SAFETY * h * pow(errmax, PSHRNK), 0.1 * h);
		}
	}

	__device__ __host__ static constexpr int sgn(float val) {
		return (0 < val) - (val < 0);
	}

	template <class T> __device__ __host__ static void odeint1(volatile T* varStart, const T b, const T q, bool shouldSavePath, float3* pathSave) {
		volatile T varScal[5];
		volatile T var[5];
		volatile T dvdz[5];
		volatile T varErr[5];
		volatile T varTemp[5];
		volatile T aks[25];
		volatile T varTmpInt[5];
		volatile T prevVar[5];

		T z = 0.0;
		T h = INITIAL_STEP_SIZE * sgn(INTEGRATION_MAX);
		T z_prev = 0;
		
		bool hit_disk = false;
		T disk_r = nanf("");
		T disk_phi = nanf("");
		T disk_redshift = 0;
		T disk_distance = 0;

		for (int i = 0; i < NUMBER_OF_EQUATIONS; i++) var[i] = varStart[i];

		bool last_theta = thetaVar > PI1_2;

		for (int nstp = 0; nstp < MAXSTP; nstp++) {

			//Save the path every ten steps for visualization if required only is included on the cpu;
			#ifndef __CUDA_ARCH__
				if (shouldSavePath && nstp % STEP_SAVE_INTERVAL == 0) {
					pathSave[nstp/ STEP_SAVE_INTERVAL] = {(float) thetaVar,(float)phiVar,(float)rVar };
				}
			#endif // __CUDA_ARCH__		

			metric::derivs(var, dvdz, b, q);
			for (int i = 0; i < NUMBER_OF_EQUATIONS; i++)
				varScal[i] = fabs(var[i]) + fabs(dvdz[i] * h) + TINY;

			rkqs(var, dvdz, z, h, varScal, b, q, varErr, varTemp, aks, varTmpInt);

			//If the step size magnitude becomes too small we are most likely very close to the black hole and we will assume we will hit it.
			if (h > MIN_STEP_SIZE || rVar <= 0) {
				break;
			}

			//If the traveled distance is large enough we can approximate space-time as flat and use these coordinates as the result
			if (z <= INTEGRATION_MAX) {
				varStart[phi_index] = var[phi_index];
				varStart[theta_index] = var[theta_index];
				varStart[r_index] = disk_redshift;
				varStart[3] = disk_r;
				varStart[4] = disk_phi;
				varStart[5] = disk_distance;
				return;
			}

			//If we want the accretion disk and the theta has crossed the 1/2pi plane we return accretion disk colors
			if (BH_USE_ACCRETION_DISK && (thetaVar > PI1_2 != last_theta) && !hit_disk) {
				float factor = (thetaVar - PI1_2) / (thetaVar - varStart[theta_index]);
				T r = (1 - factor) * rVar + factor * varStart[r_index];

				if (r > MIN_STABLE_ORBIT && r < BH_MAX_ACCRETION_RADIUS) {
				
					//Save location
					T var_disk_tmp[NUMBER_OF_EQUATIONS];
					T z_disk_tmp = z;
					for (int i = 0; i < NUMBER_OF_EQUATIONS; i++) var_disk_tmp[i] = var[i];

					//Reset vars to previous positions
					for (int i = 0; i < NUMBER_OF_EQUATIONS; i++) var[i] = varStart[i];
					z = z_prev;
					stepUntilDisk(var, dvdz, z, h, varScal, b, q, varErr, varTemp, aks, varTmpInt, varStart);
					r = varStart[r_index];
					 
					//Calculate derivatives at disk position
					metric::derivs(varStart, dvdz, b, q);
					 
					//Calculate doppler redshift
					//Accretion disk particle moves in {0,1,0} direction so cosine is simply normalized phi direction
					float3 lightdir = { dvdz[r_index], dvdz[theta_index],  dvdz[phi_index] };

					T norm = vector_ops::dot(lightdir, lightdir);
					T cos_incident_angle = rsqrt(norm) * lightdir.z;
					T orbit_speed = metric::calcSpeed<T>(r, PI1_2);
					

					disk_redshift = (1 + orbit_speed * cos_incident_angle) / sqrt(1 - sq(orbit_speed));					
					disk_distance = z;

					disk_r = varStart[r_index];
					disk_phi = varStart[phi_index];
					
					hit_disk = true;
				}							
			}
			

			last_theta = thetaVar > PI1_2;
			for (int i = 0; i < NUMBER_OF_EQUATIONS; i++) varStart[i] = var[i];
			z_prev = z;
		}

		//If we take too many steps or reached a too low step count we assume the ray hits the black hole
		varStart[theta_index] = nanf("");
		varStart[phi_index] = nanf("");

		varStart[r_index] = disk_redshift;
		varStart[3] = disk_r;
		varStart[4] = disk_phi;
		varStart[5] = disk_distance;
	};


	/// <summary>
	/// Take small steps to accuratly estimate where the ray-disk intersection is
	/// </summary>
	/// <param name="var">Array with current variables</param>
	/// <param name="dvdz">Array with derivatives at current position</param>
	/// <param name="z">Sum of all steps taken, Output is accurate length untill intersection</param>
	/// <param name="h">Current stepsize</param>
	/// <param name="varScal"></param>
	/// <param name="b"></param>
	/// <param name="q"></param>
	/// <param name="varErr"></param>
	/// <param name="varTemp"></param>
	/// <param name="aks"></param>
	/// <param name="varTmpInt"></param>
	/// <param name="varOut">Output of more accurate intersection point </param>
	/// <returns></returns>
	template <class T> __device__ __host__ void stepUntilDisk(volatile T* var, volatile  T* dvdz, T& z, T& h,
		volatile T* varScal, const T b, const T q,
		volatile T* varErr, volatile T* varTemp, volatile T* aks, volatile T* varTmpInt, volatile T* varOut) {
		bool last_theta = thetaVar > PI1_2;

		bool step_size = MIN_STEP_SIZE;
		T z_prev = z;

		while (thetaVar > PI1_2 == last_theta) {
			//Save the current position for interpolation
			for (int i = 0; i < NUMBER_OF_EQUATIONS; i++) varOut[i] = var[i];
			z_prev = z;
			//Save the position relative to the disk
			last_theta = thetaVar > PI1_2;


			//Prepare for the step
			metric::derivs(var, dvdz, b, q);
			for (int i = 0; i < NUMBER_OF_EQUATIONS; i++)
				varScal[i] = fabs(var[i]) + fabs(dvdz[i] * h) + TINY;
 			//Reduce h to go to zero near the disk to be more accurate
			h = fminf(fabs(thetaVar - PI1_2) * h, MIN_DISK_STEP_SIZE);
			//h = MIN_STEP_SIZE;
			//Take a step
			rkqs(var, dvdz, z, h, varScal, b, q, varErr, varTemp, aks, varTmpInt);
		}

		//Interpolate all vars to the disk
		float factor = (thetaVar - PI1_2) / (thetaVar - varOut[theta_index]);
		for (int i = 0; i < NUMBER_OF_EQUATIONS; i++) {
			varOut[i] = (1 - factor) * var[i] + factor * varOut[i];
		}
		z = (1 - factor) * z_prev + factor * z;

	}


	template <class T> __device__ __host__ void rkckIntegrate1(const T rV, const T thetaV, const T phiV, T* pRV,
		T* bV, T* qV, T* pThetaV, T* disk_r, T* disk_phi, bool shouldSavePath, float3* pathSave) {

		volatile T varStart[] = { rV, thetaV, phiV, *pRV, *pThetaV,0 };

		odeint1(varStart, *bV, *qV, shouldSavePath, pathSave);

		*bV = varStart[theta_index];
		*qV = varStart[phi_index];
		*pThetaV = varStart[r_index]; //Disk redshift
		*pRV = varStart[5]; //Disk distance
		*disk_r = varStart[3];
		*disk_phi = varStart[4];

		if (!isnan(varStart[theta_index])) {
			wrapToPi(*bV, *qV);
		}
		if (!isnan(varStart[3])) {
			wrapPhiToPi(*disk_phi);
		}
	}

	template <class T> __global__ void integrate_kernel(const T rV, const T thetaV, const T phiV, T* pRV,
		T* bV, T* qV, T* pThetaV, T* disk_r, T* disk_phi, int size) {
		int index = blockDim.x * blockIdx.x + threadIdx.x;

		if (index < size) {
			rkckIntegrate1(rV, thetaV, phiV, &pRV[index], &bV[index], &qV[index], &pThetaV[index],&disk_r[index],&disk_phi[index], false, nullptr);
		}
	};
}

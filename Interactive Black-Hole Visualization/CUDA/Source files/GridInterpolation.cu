#include "hip/hip_runtime.h"
#pragma once
#include "../Header files/GridInterpolation.cuh"
#include "../Header files/GridLookup.cuh"
#include "../Header files/vector_operations.cuh"
#include "../Header files/metric.cuh"
#include "../../C++/Header files/IntegrationDefines.h"
#include "../Header files/Constants.cuh"

#include <stdio.h>


__global__ void camUpdate(const float alpha, const int g, const float* camParam, float* cam) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < 7) cam[i] = (1.f - alpha) * camParam[g * 7 + i] + alpha * camParam[(g + 1) * 7 + i];
}


__global__ void pixInterpolation(const float2* viewthing, const int M, const int N, const int Gr, float2* thphi, const float2* grid,
	const int GM, const int GN, const float hor, const float ver, int* gapsave, int gridlvl,
	const float2* bhBorder, const int angleNum, const float alpha) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	int j = (blockIdx.y * blockDim.y) + threadIdx.y;
	if (i < N1 && j < M1) {
		float theta = viewthing[i * M1 + j].x + ver;
		float phi = fmodf(viewthing[i * M1 + j].y + hor + PI2, PI2);



		if (Gr > 1) {
			float2 A, B;
			float2 center = { .5f * bhBorder[0].x + .5f * bhBorder[0].y, .5f * bhBorder[1].x + .5f * bhBorder[1].y };
			float stretchRad = max(bhBorder[0].y - bhBorder[0].x, bhBorder[1].x - bhBorder[1].y) * 0.75f;
			float centerdist = (theta - center.x) * (theta - center.x) + (phi - center.y) * (phi - center.y);
			if (centerdist < stretchRad * stretchRad) {
				float angle = atan2(center.x - theta, phi - center.y);
				angle = fmodf(angle + PI2, PI2);
				int angleSlot = angle / PI2 * angleNum;

				float2 bhBorderNew = { (1.f - alpha) * bhBorder[2 * angleSlot + 2].x + alpha * bhBorder[2 * angleSlot + 3].x,
									   (1.f - alpha) * bhBorder[2 * angleSlot + 2].y + alpha * bhBorder[2 * angleSlot + 3].y };

				if (centerdist <= (bhBorderNew.x - center.x) * (bhBorderNew.x - center.x) + (bhBorderNew.y - center.y) * (bhBorderNew.y - center.y)) {
					thphi[i * M1 + j] = { -1, -1};
					return;
				}

				float tStoB = (center.x - stretchRad * sinf(angle) - bhBorderNew.x);
				float pStoB = (center.y + stretchRad * cosf(angle) - bhBorderNew.y);

				float thetaPerc = fabsf(tStoB) < 1E-5 ? 0 : 1.f - (theta - bhBorderNew.x) / tStoB;
				float phiPerc = fabsf(pStoB) < 1E-5 ? 0 : 1.f - (phi - bhBorderNew.y) / pStoB;
				float thetaA = theta - thetaPerc * (bhBorderNew.x - bhBorder[2 * angleSlot + 2].x);
				float phiA = phi - phiPerc * (bhBorderNew.y - bhBorder[2 * angleSlot + 2].y);
				float thetaB = theta - thetaPerc * (bhBorderNew.x - bhBorder[2 * angleSlot + 3].x);
				float phiB = phi - phiPerc * (bhBorderNew.y - bhBorder[2 * angleSlot + 3].y);

				A = interpolatePix<float2, true>(thetaA, phiA, M, N, 0, gridlvl, grid, GM, GN, gapsave, i, j);
				B = interpolatePix<float2, true>(thetaB, phiB, M, N, 1, gridlvl, grid, GM, GN, gapsave, i, j);
			}
			else {
				A = interpolatePix<float2, true>(theta, phi, M, N, 0, gridlvl, grid, GM, GN, gapsave, i, j);
				B = interpolatePix<float2, true>(theta, phi, M, N, 1, gridlvl, grid, GM, GN, gapsave, i, j);

			}
			if (A.x == -1 || B.x == -1) thphi[i * M1 + j] = { -1, -1};
			else {

				if (A.y < .2f * PI2 && B.y > .8f * PI2) A.y += PI2;
				if (B.y < .2f * PI2 && A.y > .8f * PI2) B.y += PI2;
				thphi[i * M1 + j] = { (1.f - alpha) * A.x + alpha * B.x, fmodf((1.f - alpha) * A.y + alpha * B.y, PI2)};
			}
		}
		else {
			float2 interpolated_tp = interpolatePix<float2,true>(theta, phi, M, N, 0, gridlvl, grid, GM, GN, gapsave, i, j);
			thphi[i * M1 + j] = interpolated_tp;
		}
	}
}

__global__ void disk_pixInterpolation(const float2* viewthing, const int M, const int N, const int Gr, float2* disk_thphi, float3* disk_incident, const float2* disk_grid, const float3* disk_incident_grid,
	const int GM, const int GN, const float hor, const float ver, int* gapsave, int gridlvl,
	const float2* bhBorder, const int angleNum, const float alpha) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	int j = (blockIdx.y * blockDim.y) + threadIdx.y;
	if (i < N1 && j < M1) {
		float theta = viewthing[i * M1 + j].x + ver;
		float phi = fmodf(viewthing[i * M1 + j].y + hor + PI2, PI2);
		if (Gr > 1) {
		
		}
		else {
			float2 interpolated_tp = interpolatePix<float2, true>(theta, phi, M, N, 0, gridlvl, disk_grid, GM, GN, gapsave, i, j);
			disk_thphi[i * M1 + j] = interpolated_tp;
			float3 interpolated_incident = interpolatePix<float3, false>(theta, phi, M, N, 0, gridlvl, disk_incident_grid, GM, GN, gapsave, i, j);
			disk_incident[i * M1 + j] = interpolated_incident;
		}


	}
}


template <class T, bool CheckPi> __device__ T interpolatePix(const float theta, const float phi, const int M, const int N, const int g, const int gridlvl,
	const T* grid, const int GM, const int GN, int* gapsave, const int i, const int j) {
	int half = (phi < PI) ? 0 : 1;
	int a = 0;
	int b = half * GM / 2;
	int gap = GM / 2;


	findBlock(theta, phi, g, grid, GM, GN, a, b, gap, gridlvl);
	gapsave[i * M1 + j] = gap;

	int k = a + gap;
	int l = b + gap;

	float factor = PI2 / (1.f * GM);
	float cornersCam[4] = { factor * a, factor * b, factor * k, factor * l };
	l = l % GM;
	T nul = { -1, -1};
	T cornersCel[12] = { grid[g * GN * GM + a * GM + b], grid[g * GN * GM + a * GM + l], grid[g * GN * GM + k * GM + b], grid[g * GN * GM + k * GM + l],
									nul, nul, nul, nul, nul, nul, nul, nul };

	

	T thphiInter = interpolateSpline<T, CheckPi>(a, b, gap, GM, GN, theta, phi, g, cornersCel, cornersCam, grid);


	return thphiInter;
}

/// <summary>
/// Interpolates the corners of a projected pixel on the celestial sky to find the position
/// of a star in the (normal, unprojected) pixel in the output image.
/// </summary>
/// <param name="t0 - t4">The theta values of the projected pixel.</param>
/// <param name="p0 - p4">The phi values of the projected pixel.</param>
/// <param name="start, starp">The star theta and phi.</param>
/// <param name="sgn">The winding order of the polygon + for CW, - for CCW.</param>
/// <returns></returns>
__device__ void interpolate(float t0, float t1, float t2, float t3, float p0, float p1, float p2, float p3,
	float& start, float& starp, int sgn, int i, int j) {
	float error = 0.00001f;

	float midT = (t0 + t1 + t2 + t3) * .25f;
	float midP = (p0 + p1 + p2 + p3) * .25f;

	float starInPixY = 0.5f;
	float starInPixX = 0.5f;

	float perc = 0.5f;
#pragma unroll
	for (int q = 0; q < 10; q++) {
		if ((fabs(start - midT) < error) && (fabs(starp - midP) < error)) break;

		float half01T = (t0 + t1) * .5f;
		float half23T = (t2 + t3) * .5f;
		float half12T = (t2 + t1) * .5f;
		float half03T = (t0 + t3) * .5f;
		float half01P = (p0 + p1) * .5f;
		float half23P = (p2 + p3) * .5f;
		float half12P = (p2 + p1) * .5f;
		float half03P = (p0 + p3) * .5f;

		float line01to23T = half23T - half01T;
		float line03to12T = half12T - half03T;
		float line01to23P = half23P - half01P;
		float line03to12P = half12P - half03P;

		float line01toStarT = start - half01T;
		float line03toStarT = start - half03T;
		float line01toStarP = starp - half01P;
		float line03toStarP = starp - half03P;

		int a = (((line03to12T * line03toStarP) - (line03toStarT * line03to12P)) > 0.f) ? 1 : -1;
		int b = (((line01to23T * line01toStarP) - (line01toStarT * line01to23P)) > 0.f) ? 1 : -1;

		perc *= 0.5f;

		if (sgn * a > 0) {
			if (sgn * b > 0) {
				t2 = half12T;
				t0 = half01T;
				t3 = midT;
				p2 = half12P;
				p0 = half01P;
				p3 = midP;
				starInPixX -= perc;
				starInPixY -= perc;
			}
			else {
				t2 = midT;
				t1 = half01T;
				t3 = half03T;
				p2 = midP;
				p1 = half01P;
				p3 = half03P;
				starInPixX -= perc;
				starInPixY += perc;
			}
		}
		else {
			if (sgn * b > 0) {
				t1 = half12T;
				t3 = half23T;
				t0 = midT;
				p1 = half12P;
				p3 = half23P;
				p0 = midP;
				starInPixX += perc;
				starInPixY -= perc;
			}
			else {
				t0 = half03T;
				t1 = midT;
				t2 = half23T;
				p0 = half03P;
				p1 = midP;
				p2 = half23P;
				starInPixX += perc;
				starInPixY += perc;
			}
		}
		midT = (t0 + t1 + t2 + t3) * .25f;
		midP = (p0 + p1 + p2 + p3) * .25f;
	}
	start = starInPixY;
	starp = starInPixX;
}

/// <summary>
/// Interpolates the location using neirest neighbour interpolation
/// </summary>
/// <param name="percDown">fraction down from top left to top right line</param>
/// <param name="percRight">fraction right from top left to bottom left line</param>
/// <param name="cornersCel">Values at top left, top right, bottom left and bottom right corners respectively</param>
/// <returns></returns>
template <class T, bool CheckPi> __device__ T interpolateNeirestNeighbour(float percDown, float percRight, T* cornersCel) {
	T corners[4] = { cornersCel[0], cornersCel[1], cornersCel[2], cornersCel[3] };

	piCheckTot<T, CheckPi>(corners, PI_CHECK_FACTOR,4);
	return corners[(int)(2 * roundf(percDown) + roundf(percRight))];
}

template <class T, bool CheckPi> __device__ T interpolateLinear(float percDown, float percRight, T* cornersCel) {
	T corners[4] = { cornersCel[0], cornersCel[1], cornersCel[2], cornersCel[3] };
	piCheckTot<T, CheckPi>(corners, PI_CHECK_FACTOR,4);

	return (1 - percRight) * ((1 - percDown) * corners[0] + percDown * corners[2]) + percRight * ((1 - percDown) * corners[1] + percDown * corners[3]);
}

template <class T> __device__ T hermite(float aValue, T& aX0, T& aX1, T& aX2, T& aX3,
	float aTension, float aBias) {
	/* Source:
	* http://paulbourke.net/miscellaneous/interpolation/
	*/

	float const v = aValue;
	float const v2 = v * v;
	float const v3 = v * v2;

	float const aa = (1.f + aBias) * (1.f - aTension) / 2.f;
	float const bb = (1.f - aBias) * (1.f - aTension) / 2.f;

	T m0 = aa * (aX1 - aX0) + bb * (aX2 - aX1);
	T m1 = aa * (aX2 - aX1) + bb * (aX3 - aX2);

	float const u0 = 2.f * v3 - 3.f * v2 + 1.f;
	float const u1 = v3 - 2.f * v2 + v;
	float const u2 = v3 - v2;
	float const u3 = -2.f * v3 + 3.f * v2;

	return u0 * aX1 + u1 * m0 + u2 * m1 + u3 * aX2;
}

template <class T, bool CheckPi> __device__ T findPoint(const int i, const int j, const int GM, const int GN, const int g,
	const int offver, const int offhor, const int gap, const T* grid, int count, T& r_check) {
	T gridpt = grid[GM * GN * g + i * GM + j];
	if (gridpt.x == -2 && gridpt.y == -2) {
		//return{ -1, -1 };
		int j2 = (j + offhor * gap + GM) % GM;
		int i2 = i + offver * gap;
		T ij2 = grid[GM * GN * g + i2 * GM + j2];
		if (ij2.x != -2 && ij2.y != -2) {

			int j0 = (j - offhor * gap + GM) % GM;
			int i0 = (i - offver * gap);

			T ij0 = grid[GM * GN * g + i0 * GM + j0];

			//If either ij0 or ij2 is not one the same surface directly return the point since there exist no visible point on the same surface 
			//if ((fabsf(ij0.z - r_check.z) < R_CHANGE_THRESHOLD) || (fabsf(ij2.z - r_check.z) < R_CHANGE_THRESHOLD)) return ij0;


			
			int jprev = (j - 3 * offhor * gap + GM) % GM;
			int jnext = (j + 3 * offhor * gap + GM) % GM;
			int iprev = i - offver * 3 * gap;
			int inext = i + offver * 3 * gap;
			if (offver != 0) {
				if (i2 == 0) {
					jnext = (j0 + GM / 2) % GM;
					inext = i0;
				}
				else if (i0 == GN - 1) {
					jprev = (j0 + GM / 2) % GM;
					iprev = i2;
				}
				else if (i2 == GN - 1) {
					inext = i0;
					jnext = (j0 + GM / 2) % GM;
				}
			}
			T ijprev = grid[GM * GN * g + iprev * GM + jprev];
			T ijnext = grid[GM * GN * g + inext * GM + jnext];

			//If the ijnext and prev are integrated and on the same surface we can use hermite interpolation.
			if (ijprev.x > -2 && ijnext.x > -2 
				//&& (fabsf(ijprev.z - r_check.z) < R_CHANGE_THRESHOLD)
				//&& (fabsf(ijnext.z - r_check.z) < R_CHANGE_THRESHOLD)
				) {
				T pt[4] = { ijprev, ij0, ij2, ijnext };
				if (pt[0].x != -1 && pt[3].x != -1) {
					piCheckTot<T, CheckPi>(pt, PI_CHECK_FACTOR, 4);
					return hermite(0.5f, pt[0], pt[1], pt[2], pt[3], 0.f, 0.f);
				}
			}

			T pt[2] = { ij2, ij0 };
			piCheckTot<T, CheckPi>(pt, PI_CHECK_FACTOR, 2);
			

			return  .5f * pt[0] +  .5f * pt[1];
		}
		else {
			//If we cant find points to interpolate with we give up on finding the point and eventually interpolated linearly.
			return { -1, -1 };
		}
	}
	return gridpt;
}

template <class T, bool CheckPi> __device__ T interpolateHermite(const int i, const int j, const int gap, const int GM, const int GN, const float percDown, const float percRight,
	const int g, T* cornersCel, const T* grid, int count, T& r_check) {

	


	int k = i + gap;
	int l = (j + gap) % GM;
	int imin1 = i - gap;
	int kplus1 = k + gap;
	int jmin1 = (j - gap + GM) % GM;
	int lplus1 = (l + gap) % GM;
	int jx = j;
	int jy = j;
	int lx = l;
	int ly = l;

	if (i == 0) {
		jx = (j + GM / 2) % GM;
		lx = (jx + gap) % GM;
		imin1 = k;
	}
	else if (k == GN - 1) {
		jy = (j + GM / 2) % GM;
		ly = (jy + gap) % GM;
		kplus1 = i;
	}

	cornersCel[4] = findPoint<T,CheckPi>(i, jmin1, GM, GN, g, 0, -1, gap, grid, count,r_check);		//4 upleft
	cornersCel[5] = findPoint<T, CheckPi>(i, lplus1, GM, GN, g, 0, 1, gap, grid, count, r_check);		//5 upright
	cornersCel[6] = findPoint<T, CheckPi>(k, jmin1, GM, GN, g, 0, -1, gap, grid, count, r_check);		//6 downleft
	cornersCel[7] = findPoint<T, CheckPi>(k, lplus1, GM, GN, g, 0, 1, gap, grid, count, r_check);		//7 downright
	cornersCel[8] = findPoint<T, CheckPi>(imin1, jx, GM, GN, g, -1, 0, gap, grid, count, r_check);		//8 lefthigh
	cornersCel[9] = findPoint<T, CheckPi>(imin1, lx, GM, GN, g, -1, 0, gap, grid, count, r_check);		//9 righthigh
	cornersCel[10] = findPoint<T, CheckPi>(kplus1, jy, GM, GN, g, 1, 0, gap, grid, count, r_check);		//10 leftdown
	cornersCel[11] = findPoint<T, CheckPi>(kplus1, ly, GM, GN, g, 1, 0, gap, grid, count, r_check);		//11 rightdown

	//If any of the extra points are in the black hole return a linear interpolation (we know the inner points are correct)
	//Or if the r coordinate differs too much meaning they are on different disk sections or in the background
	for (int q = 4; q < 12; q++) {
		if (isnan(cornersCel[q].x) || cornersCel[q].x == -1) return interpolateLinear<T, CheckPi>( percDown, percRight, cornersCel);
	}

	piCheckTot<T, CheckPi>(cornersCel, PI_CHECK_FACTOR, 12);

	T interpolateUp = hermite(percRight, cornersCel[4], cornersCel[0], cornersCel[1], cornersCel[5], 0.f, 0.f);
	T interpolateDown = hermite(percRight, cornersCel[6], cornersCel[2], cornersCel[3], cornersCel[7], 0.f, 0.f);
	T interpolateUpUp = cornersCel[8] + percRight * (cornersCel[9] - cornersCel[8]);
	T interpolateDownDown = cornersCel[10] + percRight * (cornersCel[11] - cornersCel[10]);
	//HERMITE FINITE
	return hermite(percDown, interpolateUpUp, interpolateUp, interpolateDown, interpolateDownDown, 0.f, 0.f);
}

template <class T, bool CheckPi> __device__ T interpolateSpline(const int i, const int j, const int gap, const int GM, const int GN, const float thetaCam, const float phiCam, const int g,
	T* cornersCel, float* cornersCam, const T* grid) {

	float thetaUp = cornersCam[0];
	float thetaDown = cornersCam[2];
	float phiLeft = cornersCam[1];
	float phiRight = cornersCam[3];
	

	float percDown = (thetaCam - thetaUp) / (thetaDown - thetaUp);
	float percRight = (phiCam - phiLeft) / (phiRight - phiLeft);

	


	T r_check = cornersCel[0];
	for (int q = 0; q < 4; q++) {
		if (isnan(cornersCel[q].x)) return{ nanf(""), nanf("")};
	}


	return interpolateHermite<T, CheckPi>(i, j, gap, GM, GN, percDown, percRight, g, cornersCel, grid, 0, r_check);
	//return interpolateLinear( percDown, percRight, cornersCel);
}
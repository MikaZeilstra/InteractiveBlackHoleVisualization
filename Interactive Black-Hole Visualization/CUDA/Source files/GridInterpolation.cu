#include "hip/hip_runtime.h"
#pragma once
#include "../Header files/GridInterpolation.cuh"
#include "../Header files/GridLookup.cuh"
#include "../Header files/vector_operations.cuh"
#include "../Header files/metric.cuh"
#include "../../C++/Header files/IntegrationDefines.h"
#include "../Header files/Constants.cuh"

#include <stdio.h>


__global__ void camUpdate(const float alpha, const int g, const float* camParam, float* cam) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < 7) cam[i] = (1.f - alpha) * camParam[g * 7 + i] + alpha * camParam[(g + 1) * 7 + i];
}


__global__ void pixInterpolation(const float2* viewthing, const int M, const int N, const bool should_interpolate_grids, float2* thphi, const float2* grid, const float2* grid_2,
	const int GM, const int GN, const float hor, const float ver, int* gapsave, int gridlvl,
	const float2* bhBorder, const int angleNum, const float alpha) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	int j = (blockIdx.y * blockDim.y) + threadIdx.y;


	if (i < N1 && j < M1) {
		float theta = viewthing[i * M1 + j].x + ver;
		float phi = fmodf(viewthing[i * M1 + j].y + hor + PI2, PI2);



		if (should_interpolate_grids) {
			float2 A, B;
			float2 center = { .5f * bhBorder[0].x + .5f * bhBorder[0].y, .5f * bhBorder[1].x + .5f * bhBorder[1].y };
			float stretchRad = max(bhBorder[0].y - bhBorder[0].x, bhBorder[1].x - bhBorder[1].y) * 0.75f;
			float centerdist = (theta - center.x) * (theta - center.x) + (phi - center.y) * (phi - center.y);
			if (centerdist < stretchRad * stretchRad) {
				float angle = atan2(center.x - theta, phi - center.y);
				angle = fmodf(angle + PI2, PI2);
				int angleSlot = angle / PI2 * angleNum;

				float2 bhBorderNew = { (1.f - alpha) * bhBorder[2 * angleSlot + 2].x + alpha * bhBorder[2 * angleSlot + 3].x,
									   (1.f - alpha) * bhBorder[2 * angleSlot + 2].y + alpha * bhBorder[2 * angleSlot + 3].y };

				if (centerdist <= (bhBorderNew.x - center.x) * (bhBorderNew.x - center.x) + (bhBorderNew.y - center.y) * (bhBorderNew.y - center.y)) {
					thphi[i * M1 + j] = { nanf(""), nanf("")};
					return;
				}

				float tStoB = (center.x - stretchRad * sinf(angle) - bhBorderNew.x);
				float pStoB = (center.y + stretchRad * cosf(angle) - bhBorderNew.y);

				float thetaPerc = fabsf(tStoB) < 1E-5 ? 0 : 1.f - (theta - bhBorderNew.x) / tStoB;
				float phiPerc = fabsf(pStoB) < 1E-5 ? 0 : 1.f - (phi - bhBorderNew.y) / pStoB;
				float thetaA = theta - thetaPerc * (bhBorderNew.x - bhBorder[2 * angleSlot + 2].x);
				float phiA = phi - phiPerc * (bhBorderNew.y - bhBorder[2 * angleSlot + 2].y);
				float thetaB = theta - thetaPerc * (bhBorderNew.x - bhBorder[2 * angleSlot + 3].x);
				float phiB = phi - phiPerc * (bhBorderNew.y - bhBorder[2 * angleSlot + 3].y);

				A = interpolatePix<float2, true>(thetaA, phiA, M, N, gridlvl, grid, GM, GN, gapsave, i, j);
				B = interpolatePix<float2, true>(thetaB, phiB, M, N, gridlvl, grid_2, GM, GN, gapsave, i, j);
			}
			else {
				A = interpolatePix<float2, true>(theta, phi, M, N,  gridlvl, grid, GM, GN, gapsave, i, j);
				B = interpolatePix<float2, true>(theta, phi, M, N,  gridlvl, grid_2, GM, GN, gapsave, i, j);

			}
			if (isnan(A.x) || isnan(B.x)) thphi[i * M1 + j] = { nanf(""),  nanf("") };
			else {

				if (A.y < .2f * PI2 && B.y > .8f * PI2) A.y += PI2;
				if (B.y < .2f * PI2 && A.y > .8f * PI2) B.y += PI2;
				thphi[i * M1 + j] = { (1.f - alpha) * A.x + alpha * B.x, fmodf((1.f - alpha) * A.y + alpha * B.y, PI2)};
			}
		}
		else {
			float2 interpolated_tp = interpolatePix<float2,true>(theta, phi, M, N, gridlvl, grid, GM, GN, gapsave, i, j);
			thphi[i * M1 + j] = interpolated_tp;
		}
	}
}

__global__ void disk_pixInterpolation(const float2* viewthing, const int M, const int N, const bool should_interpolate_grids, float2* disk_thphi, float3* disk_incident, const float2* disk_grid, const float3* disk_incident_grid,
	float2* disk_summary, float2* disk_summary_2, const int n_disk_angles, const int n_disk_sample, const int n_disk_segments, const int GM, const int GN, const float hor, const float ver, int* gapsave, int gridlvl,
	const float2* bhBorder, const int angleNum, const float alpha) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	int j = (blockIdx.y * blockDim.y) + threadIdx.y;
	if (i < N1 && j < M1) {
		disk_thphi[i * M1 + j] = { nanf(""),0 };

		float theta = viewthing[i * M1 + j].x + ver;
		float phi = fmodf(viewthing[i * M1 + j].y + hor + PI2, PI2);
		if (should_interpolate_grids) {
			

			//Calculate angle and angleslot from blackhole center.
			float2 center = { .5f * bhBorder[0].x + .5f * bhBorder[0].y, .5f * bhBorder[1].x + .5f * bhBorder[1].y };

			float angle = atan2(center.x - theta, phi - center.y);
			angle = fmodf(angle + PI2, PI2);
			

			center = { center.x / (float)PI2 * GM, center.y / (float)PI2 * GM };
			float2 grid_coordinates = { theta / (float)PI2 * GM, phi / (float)PI2 * GM };
			float centerdist = sqrt(vector_ops::sq_norm(grid_coordinates- center));

			int angleSlot = angle / PI2 * n_disk_angles;
			int angleSlot2 = (angleSlot + 1) % n_disk_angles;

			float angle_alpha = ((angle / PI2) * n_disk_angles) - angleSlot;


			//Find which band in the angleslot the disk falls in
			for (int segment_slot = 0; segment_slot < n_disk_segments; segment_slot++) {
				//Interpolate within each angle
				float2 interp_edges_gr_1 = (1 - angle_alpha) * disk_summary[angleSlot * (n_disk_sample + 2 * n_disk_segments) + segment_slot] +
					angle_alpha * disk_summary[angleSlot2 * (n_disk_sample + 2 * n_disk_segments) + segment_slot];


				float2 interp_edges_gr_2 = (1 - angle_alpha) * disk_summary_2[angleSlot * (n_disk_sample + 2 * n_disk_segments) + segment_slot] +
					angle_alpha * disk_summary_2[angleSlot2 * (n_disk_sample + 2 * n_disk_segments) + segment_slot];

				//Interpolate between the grids
				float2 interp_edges = (1 - alpha) * interp_edges_gr_1 + alpha * interp_edges_gr_2;

				if (centerdist > interp_edges.x && centerdist < interp_edges.y) {
					float segment_frac = (centerdist - interp_edges.x) / (interp_edges.y - interp_edges.x);

					//Calculate grid values
					float2 grid_values[] = {
						interpolate_summary(disk_summary, angle_alpha, segment_frac, segment_slot, angleSlot, angleSlot2, n_disk_segments, n_disk_sample),
						interpolate_summary(disk_summary_2,angle_alpha, segment_frac, segment_slot, angleSlot, angleSlot2, n_disk_segments, n_disk_sample)
					};

					piCheckTot<float2, true>(grid_values, PI_CHECK_FACTOR, 2);

					//Interpolate grids to get final value
					disk_thphi[i * M1 + j] = (1 - alpha) * grid_values[0] + alpha * grid_values[1];;
					return;
				}
			}
			
			
		}
		else {
			float2 interpolated_tp = interpolatePix<float2, true>(theta, phi, M, N, gridlvl, disk_grid, GM, GN, gapsave, i, j);
			disk_thphi[i * M1 + j] = interpolated_tp;
			float3 interpolated_incident = interpolatePix<float3, false>(theta, phi, M, N, gridlvl, disk_incident_grid, GM, GN, gapsave, i, j);
			disk_incident[i * M1 + j] = interpolated_incident;
		}


	}
}

/// <summary>
/// Interpolates a disksegment angleslot to give value.
/// </summary>
/// <param name="disk_summary">The disk summary to use</param>
/// <param name="segment_frac">The distance into the disk segment as fraction [0-1]</param>
/// <param name="segment_slot">The segmentslot to interpolate</param>
/// <param name="angleSlot">The anglslot to use</param>
/// <param name="n_disk_segments">number of segments per angle</param>
/// <param name="n_disk_sample">number of samples per angle</param>
/// <returns>Interpolated segment value</returns>
__device__ float2 interpolate_summary_angle(float2* disk_summary, float segment_frac, int segment_slot, int angleSlot, const int n_disk_segments, const int n_disk_sample) {
	//Get index range for this segment
	float2 index_edges = disk_summary[n_disk_segments + segment_slot + angleSlot * (n_disk_sample + 2 * n_disk_segments)];
	
	//Calculate the index segment, segment frac needs
	float index_fl = ((index_edges.y - index_edges.x) * segment_frac) + index_edges.x;

	//floor index for lower index and subtract to get the alpha
	int lower_index = index_fl;
	float index_alpha = index_fl - lower_index;

	//Get values from summary
	float2 summary_values[] = {
		disk_summary[2 * n_disk_segments + lower_index + angleSlot * (n_disk_sample + 2 * n_disk_segments)],
		disk_summary[2 * n_disk_segments + lower_index + 1 + angleSlot * (n_disk_sample + 2 * n_disk_segments)]
	};

	//Fix 2pi crossings
	piCheckTot<float2, true>(summary_values, PI_CHECK_FACTOR, 2);

	//Return linearly interpolated values
	return (1 - index_alpha) * summary_values[0] +
		(index_alpha)*summary_values[1];
}

__device__ float2 interpolate_summary(float2* disk_summary,float angle_alpha, float segment_frac, int segment_slot, int angleSlot, int angleSlot2, const int n_disk_segments, const int n_disk_sample) {

	//Calculate the values according to both adjecent angles
	float2 values[] = {
		interpolate_summary_angle(disk_summary, segment_frac, segment_slot, angleSlot, n_disk_segments, n_disk_sample),
		interpolate_summary_angle(disk_summary, segment_frac, segment_slot, angleSlot2, n_disk_segments, n_disk_sample)
	};

	//Fix 2 pi crossings
	piCheckTot<float2, true>(values, PI_CHECK_FACTOR, 2);

	//Interpolate angles to get grid value
	return (1 - angle_alpha) * values[0] + angle_alpha * values[1];

}



template <class T, bool CheckPi> __device__ T interpolatePix(const float theta, const float phi, const int M, const int N, const int gridlvl,
	const T* grid, const int GM, const int GN, int* gapsave, const int i, const int j) {
	int half = (phi < PI) ? 0 : 1;
	int a = 0;
	int b = half * GM / 2;
	int gap = GM / 2;


	findBlock(theta, phi,grid, GM, GN, a, b, gap, gridlvl);
	gapsave[i * M1 + j] = gap;

	int k = a + gap;
	int l = b + gap;

	float factor = PI2 / (1.f * GM);
	l = l % GM;
	T nul = { -1, -1};
	T cornersCel[12] = { grid[a * GM + b], grid[a * GM + l], grid[k * GM + b], grid[k * GM + l],
									nul, nul, nul, nul, nul, nul, nul, nul };

	float thetaUp = factor * a;
	float phiLeft = factor * b;
	float thetaDown = factor * k;
	float phiRight = factor * l;


	float percDown = (theta - thetaUp) / (thetaDown - thetaUp);
	float percRight = (phi - phiLeft) / (phiRight - phiLeft);

	T thphiInter = interpolateSpline<T, CheckPi>(a, b, gap, GM, GN, percDown, percRight,  cornersCel,  grid);


	return thphiInter;
}

template <class T, bool CheckPi> __device__ T interpolateGridCoord(const int GM, const int GN, T* grid, float2 grid_coord) {
	int a = grid_coord.x;
	int b = grid_coord.y;
	
	int gap = 1;
	
	while (grid[a * GM + b].x < 0) {
		gap = gap * 2;
		a = a - (a % gap);
		b = b - (b % gap);
	}
	

	int k = a + gap;
	int l = b + gap;


	l = l % GM;
	T nul = { -1, -1 };
	T cornersCel[12] = { grid[a * GM + b], grid[a * GM + l], grid[k * GM + b], grid[k * GM + l],
									nul, nul, nul, nul, nul, nul, nul, nul };

	T thphiInter = interpolateSpline<T, CheckPi>(a, b, gap, GM, GN, (grid_coord.x - a) / gap, (grid_coord.y - b) / gap, cornersCel, grid);


	return thphiInter;
}


/// <summary>
/// Interpolates the corners of a projected pixel on the celestial sky to find the position
/// of a star in the (normal, unprojected) pixel in the output image.
/// </summary>
/// <param name="t0 - t4">The theta values of the projected pixel.</param>
/// <param name="p0 - p4">The phi values of the projected pixel.</param>
/// <param name="start, starp">The star theta and phi.</param>
/// <param name="sgn">The winding order of the polygon + for CW, - for CCW.</param>
/// <returns></returns>
__device__ void interpolate(float t0, float t1, float t2, float t3, float p0, float p1, float p2, float p3,
	float& start, float& starp, int sgn, int i, int j) {
	float error = 0.00001f;

	float midT = (t0 + t1 + t2 + t3) * .25f;
	float midP = (p0 + p1 + p2 + p3) * .25f;

	float starInPixY = 0.5f;
	float starInPixX = 0.5f;

	float perc = 0.5f;
#pragma unroll
	for (int q = 0; q < 10; q++) {
		if ((fabs(start - midT) < error) && (fabs(starp - midP) < error)) break;

		float half01T = (t0 + t1) * .5f;
		float half23T = (t2 + t3) * .5f;
		float half12T = (t2 + t1) * .5f;
		float half03T = (t0 + t3) * .5f;
		float half01P = (p0 + p1) * .5f;
		float half23P = (p2 + p3) * .5f;
		float half12P = (p2 + p1) * .5f;
		float half03P = (p0 + p3) * .5f;

		float line01to23T = half23T - half01T;
		float line03to12T = half12T - half03T;
		float line01to23P = half23P - half01P;
		float line03to12P = half12P - half03P;

		float line01toStarT = start - half01T;
		float line03toStarT = start - half03T;
		float line01toStarP = starp - half01P;
		float line03toStarP = starp - half03P;

		int a = (((line03to12T * line03toStarP) - (line03toStarT * line03to12P)) > 0.f) ? 1 : -1;
		int b = (((line01to23T * line01toStarP) - (line01toStarT * line01to23P)) > 0.f) ? 1 : -1;

		perc *= 0.5f;

		if (sgn * a > 0) {
			if (sgn * b > 0) {
				t2 = half12T;
				t0 = half01T;
				t3 = midT;
				p2 = half12P;
				p0 = half01P;
				p3 = midP;
				starInPixX -= perc;
				starInPixY -= perc;
			}
			else {
				t2 = midT;
				t1 = half01T;
				t3 = half03T;
				p2 = midP;
				p1 = half01P;
				p3 = half03P;
				starInPixX -= perc;
				starInPixY += perc;
			}
		}
		else {
			if (sgn * b > 0) {
				t1 = half12T;
				t3 = half23T;
				t0 = midT;
				p1 = half12P;
				p3 = half23P;
				p0 = midP;
				starInPixX += perc;
				starInPixY -= perc;
			}
			else {
				t0 = half03T;
				t1 = midT;
				t2 = half23T;
				p0 = half03P;
				p1 = midP;
				p2 = half23P;
				starInPixX += perc;
				starInPixY += perc;
			}
		}
		midT = (t0 + t1 + t2 + t3) * .25f;
		midP = (p0 + p1 + p2 + p3) * .25f;
	}
	start = starInPixY;
	starp = starInPixX;
}

/// <summary>
/// Interpolates the location using neirest neighbour interpolation
/// </summary>
/// <param name="percDown">fraction down from top left to top right line</param>
/// <param name="percRight">fraction right from top left to bottom left line</param>
/// <param name="cornersCel">Values at top left, top right, bottom left and bottom right corners respectively</param>
/// <returns></returns>
template <class T, bool CheckPi> __device__ T interpolateNeirestNeighbour(float percDown, float percRight, T* cornersCel) {
	T corners[4] = { cornersCel[0], cornersCel[1], cornersCel[2], cornersCel[3] };

	piCheckTot<T, CheckPi>(corners, PI_CHECK_FACTOR,4);
	return corners[(int)(2 * roundf(percDown) + roundf(percRight))];
}

template <class T, bool CheckPi> __device__ T interpolateLinear(float percDown, float percRight, T* cornersCel) {
	T corners[4] = { cornersCel[0], cornersCel[1], cornersCel[2], cornersCel[3] };
	piCheckTot<T, CheckPi>(corners, PI_CHECK_FACTOR,4);

	return (1 - percRight) * ((1 - percDown) * corners[0] + percDown * corners[2]) + percRight * ((1 - percDown) * corners[1] + percDown * corners[3]);
}

template <class T> __device__ T hermite(float aValue, T& aX0, T& aX1, T& aX2, T& aX3,
	float aTension, float aBias) {
	/* Source:
	* http://paulbourke.net/miscellaneous/interpolation/
	*/

	float const v = aValue;
	float const v2 = v * v;
	float const v3 = v * v2;

	float const aa = (1.f + aBias) * (1.f - aTension) / 2.f;
	float const bb = (1.f - aBias) * (1.f - aTension) / 2.f;

	T m0 = aa * (aX1 - aX0) + bb * (aX2 - aX1);
	T m1 = aa * (aX2 - aX1) + bb * (aX3 - aX2);

	float const u0 = 2.f * v3 - 3.f * v2 + 1.f;
	float const u1 = v3 - 2.f * v2 + v;
	float const u2 = v3 - v2;
	float const u3 = -2.f * v3 + 3.f * v2;

	return u0 * aX1 + u1 * m0 + u2 * m1 + u3 * aX2;
}

template <class T, bool CheckPi> __device__ T findPoint(const int i, const int j, const int GM, const int GN, 
	const int offver, const int offhor, const int gap, const T* grid, int count, T& r_check) {
	T gridpt = grid[i * GM + j];
	
	if (gridpt.x == -2 && gridpt.y == -2) {
		//return{ -1, -1 };
		int j2 = (j + offhor * gap + GM) % GM;
		int i2 = i + offver * gap;
		T ij2 = grid[ i2 * GM + j2];
		if (ij2.x != -2 && ij2.y != -2) {

			int j0 = (j - offhor * gap + GM) % GM;
			int i0 = (i - offver * gap);

			T ij0 = grid[i0 * GM + j0];

			//If either ij0 or ij2 is not one the same surface directly return the point since there exist no visible point on the same surface 
			//if ((fabsf(ij0.z - r_check.z) < R_CHANGE_THRESHOLD) || (fabsf(ij2.z - r_check.z) < R_CHANGE_THRESHOLD)) return ij0;


			
			int jprev = (j - 3 * offhor * gap + GM) % GM;
			int jnext = (j + 3 * offhor * gap + GM) % GM;
			int iprev = i - offver * 3 * gap;
			int inext = i + offver * 3 * gap;
			if (offver != 0) {
				if (i2 == 0) {
					jnext = (j0 + GM / 2) % GM;
					inext = i0;
				}
				else if (i0 == GN - 1) {
					jprev = (j0 + GM / 2) % GM;
					iprev = i2;
				}
				else if (i2 == GN - 1) {
					inext = i0;
					jnext = (j0 + GM / 2) % GM;
				}
			}
			T ijprev = grid[iprev * GM + jprev];
			T ijnext = grid[inext * GM + jnext];

			//If the ijnext and prev are integrated and on the same surface we can use hermite interpolation.
			if (ijprev.x > -2 && ijnext.x > -2 
				//&& (fabsf(ijprev.z - r_check.z) < R_CHANGE_THRESHOLD)
				//&& (fabsf(ijnext.z - r_check.z) < R_CHANGE_THRESHOLD)
				) {
				T pt[4] = { ijprev, ij0, ij2, ijnext };
				if (pt[0].x != -1 && pt[3].x != -1) {
					piCheckTot<T, CheckPi>(pt, PI_CHECK_FACTOR, 4);
					return hermite(0.5f, pt[0], pt[1], pt[2], pt[3], 0.f, 0.f);
				}
			}

			T pt[2] = { ij2, ij0 };
			piCheckTot<T, CheckPi>(pt, PI_CHECK_FACTOR, 2);
			return  .5f * pt[0] +  .5f * pt[1];
		}
		else {
			//If we cant find points to interpolate with we give up on finding the point and eventually interpolated linearly.
			return { -1, -1 };
		}
	}
	//return { 0, 0 };
	return gridpt;
}

template <class T, bool CheckPi> __device__ T interpolateHermite(const int i, const int j, const int gap, const int GM, const int GN, const float percDown, const float percRight,
	 T* cornersCel, const T* grid, int count, T& r_check) {

	int k = i + gap;
	int l = (j + gap) % GM;
	int imin1 = i - gap;
	int kplus1 = k + gap;
	int jmin1 = (j - gap + GM) % GM;
	int lplus1 = (l + gap) % GM;
	int jx = j;
	int jy = j;
	int lx = l;
	int ly = l;

	if (i == 0) {
		jx = (j + GM / 2) % GM;
		lx = (jx + gap) % GM;
		imin1 = k;
	}
	else if (k == GN - 1) {
		jy = (j + GM / 2) % GM;
		ly = (jy + gap) % GM;
		kplus1 = i;
	}

	cornersCel[4] = findPoint<T,CheckPi>(i, jmin1, GM, GN,  0, -1, gap, grid, count,r_check);		//4 upleft
	cornersCel[5] = findPoint<T, CheckPi>(i, lplus1, GM, GN,  0, 1, gap, grid, count, r_check);		//5 upright
	cornersCel[6] = findPoint<T, CheckPi>(k, jmin1, GM, GN,  0, -1, gap, grid, count, r_check);		//6 downleft
	cornersCel[7] = findPoint<T, CheckPi>(k, lplus1, GM, GN,  0, 1, gap, grid, count, r_check);		//7 downright
	cornersCel[8] = findPoint<T, CheckPi>(imin1, jx, GM, GN,  -1, 0, gap, grid, count, r_check);		//8 lefthigh
	cornersCel[9] = findPoint<T, CheckPi>(imin1, lx, GM, GN,  -1, 0, gap, grid, count, r_check);		//9 righthigh
	cornersCel[10] = findPoint<T, CheckPi>(kplus1, jy, GM, GN,  1, 0, gap, grid, count, r_check);		//10 leftdown
	cornersCel[11] = findPoint<T, CheckPi>(kplus1, ly, GM, GN,  1, 0, gap, grid, count, r_check);		//11 rightdown

	piCheckTot<T, CheckPi>(cornersCel, PI_CHECK_FACTOR, 12);


	//If any of the extra points are in the black hole return a linear interpolation (we know the inner points are correct)
	//Or if the r coordinate differs too much meaning they are on different disk sections or in the background
	//Or if the Differnce in phi is larger than PI to indicate a change of surface of the disk
	for (int q = 4; q < 12; q++) {
		if (isnan(cornersCel[q].x) || cornersCel[q].x == -1) return interpolateLinear<T, CheckPi>(percDown, percRight, cornersCel);
		if (fabs(cornersCel[q - 1].y - cornersCel[q].y) > PI)  return interpolateLinear<T, CheckPi>(percDown, percRight, cornersCel);
	}


	T interpolateUp = hermite(percRight, cornersCel[4], cornersCel[0], cornersCel[1], cornersCel[5], 0.f, 0.f);
	T interpolateDown = hermite(percRight, cornersCel[6], cornersCel[2], cornersCel[3], cornersCel[7], 0.f, 0.f);
	T interpolateUpUp = cornersCel[8] + percRight * (cornersCel[9] - cornersCel[8]);
	T interpolateDownDown = cornersCel[10] + percRight * (cornersCel[11] - cornersCel[10]);
	//HERMITE FINITE

	T r = hermite(percDown, interpolateUpUp, interpolateUp, interpolateDown, interpolateDownDown, 0.f, 0.f);
	if(r.y < 0)  return interpolateLinear<T, CheckPi>(percDown, percRight, cornersCel);
	return r;
}

template <class T, bool CheckPi> __device__ T interpolateSpline(const int i, const int j, const int gap, const int GM, const int GN, float perc_down, float perc_right, 
	T* cornersCel, const T* grid) {


	T r_check = cornersCel[0];
	for (int q = 0; q < 4; q++) {
		if (isnan(cornersCel[q].x)) return interpolateNeirestNeighbour<T, CheckPi>(perc_down, perc_right,cornersCel);
	}

	return interpolateHermite<T, CheckPi>(i, j, gap, GM, GN, perc_down, perc_right, cornersCel, grid, 0, r_check);
	//return interpolateLinear<T, CheckPi>(perc_down, perc_right, cornersCel);
}